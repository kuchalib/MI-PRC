#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS
#include "hip/hip_runtime.h"
#include ""

#include "md5.c"

#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>
#include <ctype.h>
#include <math.h>


static void HandleError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define THREADS 128 /// 128
#define BLOCKS 32 /// 32

#define THRESHOLD 1000
#define DICTIONARY_THRESHOLD 250
#define MAX_ALPHABET_SIZE 95
#define MAX_RULES_COUNT 5

#define MAX_WORD_LENGTH 200
#define GRANULARITY 5000
#define MEMORY_RATIO 2



#define ADD_BACK 1

#define DEVICE 0 // toto mozna bude nutne menit na STARU!

char tnumbers[11] = "0123456789";
char tlowercase[27] = "abcdefghijklmnopqrstuvwxyz";
char tuppercase[27] = "ABCDEFGHIJKLMNOPQRTSUVWXYZ";
char tlowerupper[53] = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRTSUVWXYZ";
char tloweruppernums[63] = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRTSUVWXYZ0123456789";
char allchars[95] = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRTSUVWXYZ0123456789!\"#$%&'()*+,-./:;<=>?@[\\]^_`{|}~";
char * alph[6] = { &tnumbers[0], &tlowercase[0], &tuppercase[0], &tlowerupper[0], &tloweruppernums[0], allchars };
char sizes[6] = { 10, 26, 26, 52, 62, 94 };
int limits[6] = { 19, 13, 13, 11, 10, 9 }; // max word length

char * valuePlaceholder;

__constant__ char alphabetGPU[MAX_ALPHABET_SIZE];
__constant__ uint32_t hashGPU[4]; 

__constant__ char dictionaryAlphabetGPU[MAX_RULES_COUNT * MAX_ALPHABET_SIZE];
__constant__ int dictionarySizeGPU[MAX_RULES_COUNT];
__constant__ int minLengthGPU[MAX_RULES_COUNT];
__constant__ int maxLengthGPU[MAX_RULES_COUNT];
__constant__ int rulesGPU[MAX_RULES_COUNT];

int threads = THREADS;
int blocks = BLOCKS; 


#pragma region GPU
__device__ void isHashEqualNewDevice(uint32_t * hash1, uint32_t * hash2, bool * ret)
{
	*ret = true;
	for (int i = 0; i < 4; i++)
	{
		if (hash1[i] != hash2[i])
		{
			*ret = false;
			return;
		}
	}
}

#pragma region Brute force
__device__ void * bruteForceStepDevice(int stringLength, int alphabetSize, char * alphabet, char * text, int * initialPermutation, uint64_t count, char * valuePlaceholder, char * textStartAddress, int totalStringLength)
{
	// pouzit uint32_t

	uint32_t hashPlaceHolderNew[4];

	uint32_t hashLocal[4]; 
	memcpy(hashLocal, hashGPU, 4 * sizeof(uint32_t));
	for (int i = 0; i < stringLength; i++)
	{
		// nastaveni stringu do pocatecniho stavu
		text[i] = alphabet[initialPermutation[i]];
	}


	text[stringLength] = 0;
	bool overflow = false;
	uint64_t localCount = 0;
	bool retTmp = false;
	uint8_t msg[200]; 
	while (!overflow)
	{
		//printf("Zkousim %s\n", textStartAddress); 
		md5Device(textStartAddress, totalStringLength, hashPlaceHolderNew, msg);
		isHashEqualNewDevice(hashPlaceHolderNew, hashLocal, &retTmp);
		if (retTmp)
		{
			memcpy(valuePlaceholder, textStartAddress, totalStringLength * sizeof(char));
			//free(hashedString);
			return; 
		}
		initialPermutation[0]++;
		initialPermutation[0] %= alphabetSize;
		text[0] = alphabet[initialPermutation[0]];
		if (initialPermutation[0] == 0)
		{
			for (int i = 1; i < stringLength; i++)
			{
				// carry chain
				initialPermutation[i]++;
				initialPermutation[i] %= alphabetSize;
				text[i] = alphabet[initialPermutation[i]];
				if (initialPermutation[i] != 0)
					break;
				else
					if (i == stringLength)
						overflow = true;
			}
		}

		localCount++;
		if (localCount >= count)
			break;
		if (localCount % THRESHOLD == 0)
		{
			if (valuePlaceholder[0] != 0)
				return; 
		}
	}
	// hash nenalezen
	return NULL;
}

__global__ void bruteForceDevice(int len, uint64_t total, int alphabetSize, char * valuePlaceholder)
{
	hipError_t cudaStatus;
	char * text;
	int * initialPermutation; 
	int threadCount = blockDim.x;
	int blockCount = gridDim.x;
	double tmp = ((double)total / blockCount);
	uint64_t blockWork = (uint64_t)ceil(tmp);
	tmp = (double)total / (blockCount*threadCount);
	uint64_t threadWork = (uint64_t)ceil(tmp);
	uint64_t start = blockIdx.x * blockWork + threadIdx.x * threadWork;
	

	text = (char*)malloc(len + 1); 

	if (text == 0)
	{
		printf("bruteForceDevice, nelze alokovat pamet pro text\n");
		return;
	}

	initialPermutation = (int*)malloc(len * sizeof(int));
	if (initialPermutation == NULL) {
		printf("bruteForceDevice, nelze alokovat pamet\n");
		return;
	}
	

	for (int i = 0; i < len; i++)
	{

		initialPermutation[i] = start % alphabetSize;
		start /= alphabetSize; 
	}
	bruteForceStepDevice(len, alphabetSize, alphabetGPU, text, initialPermutation, threadWork, valuePlaceholder, text, len);

	free(text);
	free(initialPermutation);
}

__host__ char * cudaBruteForceStart(int minLength, int maxLength, char * alphabet, int alphabetSize, uint32_t hash[4])
{
	hipError_t cudaStatus;
	char * originalValue = NULL;
	HANDLE_ERROR(hipMemcpyToSymbol((const void *)alphabetGPU, alphabet, alphabetSize, 0, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToSymbol((const void *)hashGPU, hash, sizeof(uint32_t) * 4, 0, hipMemcpyHostToDevice));

	for (int i = minLength; i <= maxLength; i++)
	{
		originalValue = NULL; 
		HANDLE_ERROR(hipMalloc((void**)&valuePlaceholder, (size_t)(i * sizeof(char) + 1)));

		HANDLE_ERROR(hipMemset(valuePlaceholder, 0, (size_t)i + 1));
		uint64_t total = pow(alphabetSize, i);
		bruteForceDevice << <blocks, threads >> >(i, total, alphabetSize, valuePlaceholder);
		originalValue = (char *)malloc(i * sizeof(char) + 1);
		hipDeviceSynchronize();

		HANDLE_ERROR(hipMemcpy(originalValue, valuePlaceholder, i, hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipFree(valuePlaceholder));

		if (originalValue[0] != 0)
		{
			originalValue[i] = 0; 
			break;
		}

		free(originalValue);
	}
	return originalValue; 
}

#pragma endregion Brute force

// memlimits in bytes!!!
__host__ char * prepareWords(FILE * fp, size_t memLimit, unsigned int * count, int * maxLen, bool * eof)
{
	long start = ftell(fp); 

	register int _maxLen = 0;
	register unsigned int _count = 0;
	char bigbuffer[5000];
	char word[MAX_WORD_LENGTH];
	bool _eof = false; 
	int index = 0; 
	while (true)
	{
		for (int i = 0; i < GRANULARITY; i++)
		{
			if (fscanf(fp, "%s", bigbuffer) == EOF)
			{
				_eof = true;
				break; 
			}
			int tmp = strlen(bigbuffer);
			if (tmp > MAX_WORD_LENGTH)
				continue; 
			_maxLen = max(tmp, _maxLen); 
			_count++; 
			index++; 
		}
		_maxLen += 2;
		size_t tmp = _maxLen * _count * sizeof(char); 
		if (tmp > memLimit)
		{
			_count -= index;
			break; 
		}
		if (_count > UINT_MAX - 2 * GRANULARITY)
			break; 
		if (_eof)
			break; 
		index = 0; 
	}

	*eof = _eof; 
	fseek(fp, start, SEEK_SET); 
	long tmp = ftell(fp); 
	char * words = (char*)malloc(_count * _maxLen * sizeof(char));
	for (unsigned int i = 0; i < _count; i++)
	{ 
		fscanf(fp, "%s", words + i*_maxLen);
		words[(i + 1)*_maxLen - 1] = (char)strlen(words + i*_maxLen);
	}

	*count = _count;
	*maxLen = _maxLen;
	return words; 


	
}

__global__ void DictionaryAttackStep(unsigned int count, char * words, char * valuePlaceholder, int maxLen, int rulesCount)
{
	int threadCount = blockDim.x;
	int blockCount = gridDim.x;
	char word[MAX_WORD_LENGTH + 40]; 
	uint8_t msg[300];
	uint32_t hashPlaceHolderNew[4];
	int initialPermutation[25]; 

	bool retTmp = false; 

	hipError_t cudaStatus;
	uint32_t hashLocal[4];
	memcpy(hashLocal, hashGPU, 4 * sizeof(uint32_t));

	double tmp = ((double)count / blockCount);
	uint64_t blockWork = (uint64_t)ceil(tmp);
	tmp = (double)count / (blockCount*threadCount);
	uint64_t threadWork = (uint64_t)ceil(tmp);

	int start = blockIdx.x * blockWork + threadIdx.x * threadWork;
	int end = start + threadWork;
	char * address = words + start*maxLen;
	char * endAddress = words + end*maxLen; 
	char * realEndAddress = words + count * maxLen; 

	endAddress > realEndAddress ? endAddress = realEndAddress : endAddress = endAddress; 
	for (int i = 0; address < endAddress; address += maxLen, i++)
	{
		void * destination = memcpy(word, address, maxLen);
		size_t len = (size_t)((unsigned char)word[maxLen - 1]);
		md5Device(word, len, hashPlaceHolderNew, msg);
		isHashEqualNewDevice(hashPlaceHolderNew, hashLocal, &retTmp);
		if (retTmp)
		{
			memcpy(valuePlaceholder, word, len + 1);
			//free(hashedString);
			return;
		}
		for (int j = 0; j < rulesCount; j++)
		{
			if (rulesGPU[j] == ADD_BACK)
			{
				for (int k = minLengthGPU[j]; k <= maxLengthGPU[j]; k++)
				{
					char * editStart = word + len;
					int finalLength = len + k;
					editStart[k] = 0; // vynulovat konec stringu
					for (int l = 0; l < k; l++)
					{
						initialPermutation[l] = 0; 
					}
					uint64_t count = (uint64_t)pow((double)dictionarySizeGPU[j], (double)k);
					//printf("%ddsdsdsd %c\n", count, dictionaryAlphabetGPU[5]);
					bruteForceStepDevice(k, dictionarySizeGPU[j], dictionaryAlphabetGPU + j*MAX_ALPHABET_SIZE, editStart, initialPermutation, count , valuePlaceholder, word, finalLength);

				}
			}
		}

		if (i == DICTIONARY_THRESHOLD)
		{
			if (valuePlaceholder[0] != 0)
				return;
			i = 0; 
		}

	}

}

__host__ char * cudaDictionaryAttack(char * dictionaryFile, uint32_t hash[4], char ** alphabet, int * alphabetSize, int * minLength, int * maxLength, int * rules, int rulesCount)
{

	// GPU MEMORY:
	char * usedMemory, *tmpMemory;
	char * originalValue = NULL;
	bool retTmp = false;
	char word[MAX_WORD_LENGTH]; 
	bool eof = false; ;
	int maxLen = -1, lastLen;
	unsigned int count = 0, last_count; 
	size_t * freeMemory,*totalMemory; 
	size_t memLimit = 0;
	bool __eof = false; 
	uint64_t nacteno = 0; 

	FILE * fp = fopen(dictionaryFile, "r");
	if (fp == NULL)
	{
		printf("Cant open the dictionary");
		return NULL;
	}
	
	if (rulesCount > 0 && rulesCount <= MAX_RULES_COUNT)
	{
		int * _rules = (int *)malloc(rulesCount * sizeof(int));
		int * _alphabetSize = (int *)malloc(rulesCount * sizeof(int));
		int * _minLength = (int *)malloc(rulesCount * sizeof(int));
		int * _maxLength = (int *)malloc(rulesCount * sizeof(int));
		char * _alphabet = (char *)malloc(rulesCount * MAX_ALPHABET_SIZE * sizeof(char));

		for (int i = 0; i < rulesCount; i++)
		{
			_rules[i] = rules[i];
			_alphabetSize[i] = alphabetSize[i];
			_minLength[i] = minLength[i];
			_maxLength[i] = maxLength[i];
			memcpy(_alphabet + i*MAX_ALPHABET_SIZE, alphabet[i], alphabetSize[i]);
		}
		
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(rulesGPU), _rules, rulesCount*sizeof(int), 0, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dictionarySizeGPU), _alphabetSize, rulesCount * sizeof(int), 0, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL( minLengthGPU), _minLength, rulesCount * sizeof(int), 0, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(maxLengthGPU), _maxLength, rulesCount * sizeof(int), 0, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dictionaryAlphabetGPU), _alphabet, rulesCount * MAX_ALPHABET_SIZE * sizeof(char), 0, hipMemcpyHostToDevice));
		free(_rules);
		free(_alphabetSize);
		free(_minLength);
		free(_maxLength);
		free(_alphabet); 
	}
	hipDeviceProp_t prop; 
	HANDLE_ERROR(hipGetDeviceProperties(&prop, DEVICE)); 
	memLimit = prop.totalGlobalMem / MEMORY_RATIO;

	char * words = prepareWords(fp, memLimit, &count, &maxLen, &__eof);
	nacteno += count; 
	HANDLE_ERROR(hipMemcpyToSymbol((const void *)hashGPU, hash, sizeof(uint32_t) * 4, 0, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMalloc(&usedMemory, maxLen * count * sizeof(char))); 
	HANDLE_ERROR(hipMemcpy(usedMemory,words, maxLen * count * sizeof(char), hipMemcpyHostToDevice));
	free(words); 
	
	do 
	{
		eof = __eof; 
		last_count = count; 
		lastLen = maxLen;
		originalValue = (char *)malloc(maxLen * sizeof(char));
		HANDLE_ERROR(hipMalloc((void**)&valuePlaceholder, (size_t)(maxLen * sizeof(char))));
		HANDLE_ERROR(hipMemset(valuePlaceholder, 0, (size_t)maxLen * sizeof(char)));
		if (count == 0)
			break; 

		DictionaryAttackStep << < blocks, threads >> > (count, usedMemory, valuePlaceholder, maxLen, rulesCount); 
		// run kernel
		words = prepareWords(fp, memLimit, &count, &maxLen, &__eof);
		nacteno += count;
		if (count == 0 && !eof)
		{
			printf("Too little VRAM! Exiting now\n");
			return NULL; 
		}
		if (count > 0)
		{
			HANDLE_ERROR(hipMalloc(&tmpMemory, maxLen * count * sizeof(char)));
			HANDLE_ERROR(hipMemcpy(tmpMemory, words, maxLen * count * sizeof(char), hipMemcpyHostToDevice));
			free(words);
		}
		hipDeviceSynchronize();
		HANDLE_ERROR(hipFree(usedMemory));
		usedMemory = tmpMemory; 

		HANDLE_ERROR(hipMemcpy(originalValue, valuePlaceholder, lastLen, hipMemcpyDeviceToHost));

		if (originalValue[0] != 0)
			break; 
		//free(originalValue); 
		HANDLE_ERROR(hipFree(valuePlaceholder));
	} while (!eof); 
	
	fclose(fp); 
	if (originalValue[0] == 0)
		return NULL;
	else
		return originalValue; 
	

}

#pragma endregion GPU

#pragma region CPU
void isHashEqualNew(uint32_t * hash1, uint32_t * hash2, bool * ret)
{
	*ret = true; 
	for (int i = 0; i < 4; i++)
	{
		if (hash1[i] != hash2[i])
		{
			*ret = false;
			return; 
		}
	}
}

// nerekurzivni volani - idealni pro GPU?
// initialPermutation nastavi pocatecni kombinaci pismen, toto se zjisti tak, ze se vezme pocatecni hodnota pro dane vlakno a postupne se vymoduli/vydeli toto cislo a ziska se tim permutace. 
//Bude to fungovat podobne jako kdyz se napr. desitkove cislo prevadi na sestnactkove, count urcuje pocet iteraci
// stringLength - delka editovaneho textu
// totalStringLength - celkova delka textu
// text - zacatek editace
// textStartAddress - pocatecni adresa textu
char * bruteForceStepNew(int stringLength, char * alphabet, int alphabetSize, char * text, uint32_t hash[4], int * initialPermutation, uint64_t count, char * textStartAddress, int totalStringLength)
{
	// pouzit uint32_t
	uint32_t hashPlaceHolderNew[4]; 
	for (int i = 0; i < stringLength; i++)
	{
		// nastaveni stringu do pocatecniho stavu
		text[i] = alphabet[initialPermutation[i]];
	}
	text[stringLength] = 0;
	bool overflow = false;
	double localCount = 0;
	bool retTmp = false; 

	while (!overflow)
	{
		md5(textStartAddress, totalStringLength, hashPlaceHolderNew);
		isHashEqualNew(hashPlaceHolderNew, hash, &retTmp);
		if (retTmp)
		{
			//free(hashedString);
			return text;
		}
		initialPermutation[0]++;
		initialPermutation[0] %= alphabetSize;
		text[0] = alphabet[initialPermutation[0]];
		if (initialPermutation[0] == 0)
		{
			for (int i = 1; i < stringLength; i++)
			{
				// carry chain
				initialPermutation[i]++;
				initialPermutation[i] %= alphabetSize;
				text[i] = alphabet[initialPermutation[i]];
				if (initialPermutation[i] != 0)
					break;
				else
					if (i == stringLength)
						overflow = true;
			}
		}

		localCount++;
		if (localCount >= count)
			break;
	}
	// hash nenalezen
	return NULL;
}

char * bruteForceNew(int minLength, int maxLength, char * alphabet, int alphabetSize, uint32_t hash[4])
{
	for (int i = minLength; i <= maxLength; i++)
	{
		char * text = (char *)malloc((i + 1) * sizeof(char));
		//char * value = bruteForceStepRec(0, i, alphabet, alphabetSize, text, hash);
		int * initialPermutation = (int*)calloc(sizeof(int), i);
		uint64_t count = pow(alphabetSize, i);
		char * value = bruteForceStepNew(i, alphabet, alphabetSize, text, hash, initialPermutation, count, text, i);
		if (value != NULL)
			return value;
		free(text);
	}

	return NULL;
}

// Rozsireny slovnikovy utok, aplikuje pouze pravidla pridavani pismen v rozsahu minLength - maxLength za slovo, v budoucnu lze upravovat pomoci rules
char * dictionaryAttack(char * dictionaryFile, uint32_t hash[4], char ** alphabet, int * alphabetSize, int * minLength, int * maxLength, int * rules, int rulesCount)
{
	uint32_t hashPlaceholder[4];
	FILE * fp = fopen(dictionaryFile, "r");
	bool retTmp = false;
	int maxLen = 0;
	if (fp == NULL)
	{
		printf("Cant open the dictionary");
		return NULL;
	}
	char * word = (char*)malloc(255);
	while (true)
	{
		
		if (fscanf(fp, "%s", word) == EOF)
			break;
		int len = strlen(word);
		maxLen = max(maxLen, len); 
		md5(word, len, hashPlaceholder);
		isHashEqualNew(hashPlaceholder, hash, &retTmp);
		if (retTmp)
		{
			//free(hashedString);
			fclose(fp);
			return word;
		}
		
		for (int i = 0; i < rulesCount; i++)
		{
			if (rules[i] == ADD_BACK)
			{
				for (int j = minLength[i]; j <= maxLength[i]; j++)
				{
					char * editStart = word + len;
					int finalLength = len + j; 
					editStart[j] = 0; // vynulovat konec stringu
					int * initialPermutation = (int*)calloc(j, sizeof(int)); 
					uint64_t count = pow(alphabetSize[i], j);
					char * result = bruteForceStepNew(j, alphabet[i], alphabetSize[i], editStart, hash, initialPermutation, count, word, finalLength); 
					if (result != NULL)
					{
						fclose(fp);
						return word;
					}
					// free(initialPermutation); 
				}
			}
		}
	}

	free(word);
	fclose(fp);
	return NULL;
}


void badUsage()
{
	printf("Usage: PATH_TO_PROGRAM mode [path_to_dictionary | alphabet] hash [min_length] [max_length]\n\n");
	printf("MODE:\n");
	printf("0 - Dictionary attack\n");
	printf("    Usage: PATH_TO_PROGRAM 0 path_to_dictionary hash {0-%d}[[rule] [alphabet] [min_length] [max_length]]\n", MAX_RULES_COUNT);
	printf("1 - Brute-force attack\n");
	printf("    Usage: PATH_TO_PROGRAM 1 alphabet hash min_length max_length\n");
	printf("2 - Brute-force attack GPU\n");
	printf("    Usage: PATH_TO_PROGRAM 1 alphabet hash min_length max_length blocks threads\n");
	printf("3 - Dictionary attack - GPU\n");
	printf("    Usage: PATH_TO_PROGRAM 0 path_to_dictionary hash {0-%d}[[rule] [alphabet] [min_length] [max_length]] blocks threads\n", MAX_RULES_COUNT);
	printf("ALPHABET:\n");
	printf("0 - numbers only\n");
	printf("1 - lower case\n");
	printf("2 - upper case\n");
	printf("3 - lower+upper case\n");
	printf("4 - lower+upper case + numbers\n");
	printf("5 - all characters\n");

}



uint32_t * stringToHashNew(char * hashString)
{
	uint8_t * hashArray = (uint8_t *)malloc(16 * sizeof(uint8_t));
	uint32_t * hashNew = (uint32_t *)calloc(4, sizeof(uint32_t));
	for (int i = 0; i < 16; i++)
	{
		uint8_t hiNibble = (uint8_t)tolower(hashString[2 * i]);
		hiNibble > 57 ? hiNibble -= 87 : hiNibble -= 48;
		uint8_t loNibble = (uint8_t)tolower(hashString[2 * i + 1]);
		loNibble > 57 ? loNibble -= 87 : loNibble -= 48;
		hashArray[i] = hiNibble << 4 | loNibble;
	}
	for (int i = 0; i < 16; i++)
	{
		uint32_t tmp = hashArray[i] << ((i % 4) * 8); 
		hashNew[i / 4] |= tmp; 
	}
	free(hashArray);
	return hashNew;
}



#pragma endregion CPU

int main(int argc, char *argv[])
{
 // args 1 0 52c69e3a57331081823331c4e69d3f2e 6 6 (999999)
 // 0 E:\\Dictionary\slovnik.txt 1b34d880de0281139ed8d526b9462e9d 1 1 1 3
// 0 E:\\words.txt 77360f71a0c28c212111a617b90466d8 0 1 1 3
// 3 E:\\Dictionary\slovnik.txt b8074d446492705a6dd7d5e75aaf954f 1 0 1 1 32 128

	uint32_t *hash;
	char *originalString;
	int mode = -1;
	int alphabetMode = -1;
	char *_alphabet;
	int alphabetLen;

	if (argc < 4) {
		badUsage();
		return -1;
	}

	mode = atoi(argv[1]);
	hash = stringToHashNew(argv[3]);

	if (mode == 0 || mode == 3) {
		//	printf("    Usage: PATH_TO_PROGRAM 0 path_to_dictionary hash {0-%d}[[rule] [alphabet] [min_length] [max_length]]\n", MAX_RULES_COUNT);
		char * alphabet[MAX_RULES_COUNT];
		int alphabetSize[MAX_RULES_COUNT];
		int minLength[MAX_RULES_COUNT];
		int maxLength[MAX_RULES_COUNT]; 
		int rules[MAX_RULES_COUNT];
		int rulesCount = 0; 
		if (argc > 4)
		{
			int argcTmp = argc; 
			if (mode == 3)
				argcTmp -= 2; 
			if (argcTmp % 4 == 0)
			{
				
				for (int i = 4; i < argcTmp; i += 4)
				{
					if (rulesCount + 1 == MAX_RULES_COUNT)
					{
						printf("Too many rules\n");
						break; 
					}
					rules[rulesCount] = atoi(argv[i]);
					alphabetMode = atoi(argv[i + 1]);
					if (alphabetMode >= 0 && alphabetMode <= 5) {
						alphabet[rulesCount] = alph[alphabetMode];
						alphabetSize[rulesCount] = sizes[alphabetMode];
					}

					else
					{
						printf("Incorrectly defined dictionary rules - ignore all\n");
						rulesCount = 0; 
						break; 
					}
					minLength[rulesCount] = atoi(argv[i + 2]);
					maxLength[rulesCount] = atoi(argv[i + 3]); 
					rulesCount++; 
				}
			}
			else
				printf("Incorrectly defined dictionary rules - ignore all\n");
		}
		if(mode == 0)
			originalString = dictionaryAttack(argv[2], hash, alphabet, alphabetSize, minLength, maxLength, rules, rulesCount);
		else if (mode == 3)
		{
			if (argc < rulesCount * 4 + 6)
			{
				badUsage();
				return -1; 
			}
			blocks = atoi(argv[rulesCount * 4 + 4]);
			threads = atoi(argv[rulesCount * 4 + 5]);
			originalString = cudaDictionaryAttack(argv[2], hash, alphabet, alphabetSize, minLength, maxLength, rules, rulesCount);
		}
		if (originalString == NULL) {
			printf("No matches!\n");
			return 0;
		}
		else if (originalString[0] == '0')
		{
			printf("No matches!\n");
			return 0;
		}
		else {
			printf("%s\n", originalString);
			return 0;
		}
	}
	if (mode == 1) {
		if (argc < 6) {
			badUsage();
			return -1;
		}

		int minLenght = -1, maxLength = -1;
		alphabetMode = atoi(argv[2]);
		minLenght = atoi(argv[4]);
		maxLength = atoi(argv[5]);

		if (minLenght > maxLength) {
			printf("Minimum length must be less than or equal to the maximum length.\n");
			return -1;
		}

		if (alphabetMode >= 0 && alphabetMode <= 5) {
			_alphabet = alph[alphabetMode];
			alphabetLen = sizes[alphabetMode];
		}
		else {
			badUsage();
			return -1;
		}

		//originalString = cudaBruteForceStart(minLenght, maxLength, _alphabet, alphabetLen, hash);
		originalString = bruteForceNew(minLenght, maxLength, _alphabet, alphabetLen, hash);
		if (originalString == NULL) {
			printf("No matches!\n");
			return 0;
		}
		else {
			printf("%s\n", originalString);
			return 0;
		}
		
	}
	else if (mode == 2)
	{
		if (argc < 8) {
			badUsage();
			return -1;
		}

		int minLenght = -1, maxLength = -1;
		alphabetMode = atoi(argv[2]);
		minLenght = atoi(argv[4]);
		maxLength = atoi(argv[5]);
		blocks = atoi(argv[6]);
		threads = atoi(argv[7]); 

		if (minLenght > maxLength) {
			printf("Minimum length must be less than or equal to the maximum length.\n");
			return -1;
		}

		if (alphabetMode >= 0 && alphabetMode <= 5) {
			_alphabet = alph[alphabetMode];
			alphabetLen = sizes[alphabetMode];
		}
		else {
			badUsage();
			return -1;
		}

		originalString = cudaBruteForceStart(minLenght, maxLength, _alphabet, alphabetLen, hash);
		//originalString = bruteForceNew(minLenght, maxLength, _alphabet, alphabetLen, hash);
		if (originalString[0] == 0) {
			printf("No matches!\n");
			return 0;
		}
		else {
			printf("%s\n", originalString);
			return 0;
		}
	}
	return 0;
}


